
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <string.h>

/* 
 * Reference code from Colin Percival's original version.  (One tweak: for loop
 * changed to be ++ instead of += 2, as that was unnecessary).
 * Used for validation of cuda implementation.
 */

static inline void xor_salsa8_original(uint32_t B[16], const uint32_t Bx[16])
{
	uint32_t x00,x01,x02,x03,x04,x05,x06,x07,x08,x09,x10,x11,x12,x13,x14,x15;
	int i;

	x00 = (B[ 0] ^= Bx[ 0]);
	x01 = (B[ 1] ^= Bx[ 1]);
	x02 = (B[ 2] ^= Bx[ 2]);
	x03 = (B[ 3] ^= Bx[ 3]);
	x04 = (B[ 4] ^= Bx[ 4]);
	x05 = (B[ 5] ^= Bx[ 5]);
	x06 = (B[ 6] ^= Bx[ 6]);
	x07 = (B[ 7] ^= Bx[ 7]);
	x08 = (B[ 8] ^= Bx[ 8]);
	x09 = (B[ 9] ^= Bx[ 9]);
	x10 = (B[10] ^= Bx[10]);
	x11 = (B[11] ^= Bx[11]);
	x12 = (B[12] ^= Bx[12]);
	x13 = (B[13] ^= Bx[13]);
	x14 = (B[14] ^= Bx[14]);
	x15 = (B[15] ^= Bx[15]);
	for (i = 0; i < 4; i++) {
#define R(a, b) (((a) << (b)) | ((a) >> (32 - (b))))
		/* Operate on columns. */
		x04 ^= R(x00+x12, 7);	x09 ^= R(x05+x01, 7);
		x14 ^= R(x10+x06, 7);	x03 ^= R(x15+x11, 7);

		x08 ^= R(x04+x00, 9);	x13 ^= R(x09+x05, 9);
		x02 ^= R(x14+x10, 9);	x07 ^= R(x03+x15, 9);
		
		x12 ^= R(x08+x04,13);	x01 ^= R(x13+x09,13);
		x06 ^= R(x02+x14,13);	x11 ^= R(x07+x03,13);
		
		x00 ^= R(x12+x08,18);	x05 ^= R(x01+x13,18);
		x10 ^= R(x06+x02,18);	x15 ^= R(x11+x07,18);

		/* Operate on rows. */
		x01 ^= R(x00+x03, 7);	x06 ^= R(x05+x04, 7);
		x11 ^= R(x10+x09, 7);	x12 ^= R(x15+x14, 7);
		
		x02 ^= R(x01+x00, 9);	x07 ^= R(x06+x05, 9);
		x08 ^= R(x11+x10, 9);	x13 ^= R(x12+x15, 9);
		
		x03 ^= R(x02+x01,13);	x04 ^= R(x07+x06,13);
		x09 ^= R(x08+x11,13);	x14 ^= R(x13+x12,13);
		
		x00 ^= R(x03+x02,18);	x05 ^= R(x04+x07,18);
		x10 ^= R(x09+x08,18);	x15 ^= R(x14+x13,18);
#undef R
	}
	B[ 0] += x00;
	B[ 1] += x01;
	B[ 2] += x02;
	B[ 3] += x03;
	B[ 4] += x04;
	B[ 5] += x05;
	B[ 6] += x06;
	B[ 7] += x07;
	B[ 8] += x08;
	B[ 9] += x09;
	B[10] += x10;
	B[11] += x11;
	B[12] += x12;
	B[13] += x13;
	B[14] += x14;
	B[15] += x15;
}

/* Modifies X in place to produce scrypt(X) */

void scrypt_core_original(uint32_t *X) 
{
  uint32_t V[32*1024];

  for (int i = 0; i < 1024; i++) {
    memcpy(&V[i*32], X, 128);
    xor_salsa8_original(&X[0], &X[16]);
    xor_salsa8_original(&X[16], &X[0]);
  }
  for (int i = 0; i < 1024; i++) {
    int j = 32 * (X[16] & 1023);
    for (int k = 0; k < 32; k++) 
      X[k] ^= V[j + k];
    xor_salsa8_original(&X[0], &X[16]);
    xor_salsa8_original(&X[16], &X[0]);
  }
}
